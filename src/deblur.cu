#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

// Kernel Functions for Deconvolution
__global__ void complexMul(hipComplex *A, hipComplex *B, hipComplex *C)
{
    unsigned int i = blockIdx.x * gridDim.y * gridDim.z *
                      blockDim.x + blockIdx.y * gridDim.z *
                      blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
    C[i] = hipCmulf(A[i], B[i]);
}

__global__ void floatDiv(float *A, float *B, float *C)
{
    unsigned int i = blockIdx.x * gridDim.y * gridDim.z *
                      blockDim.x + blockIdx.y * gridDim.z *
                      blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
    C[i] = A[i] / B[i];
}

__global__ void floatMul(float *A, float *B, float *C)
{
    unsigned int i = blockIdx.x * gridDim.y * gridDim.z *
                      blockDim.x + blockIdx.y * gridDim.z *
                      blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
    C[i] = A[i] * B[i];
}

/* Parameters:
  nIter = Number of Iterations
  N1 = size of Dim 1
  N2 = size of Dim 2
  N3 = size of Dim 3
  *hImage = pointer to image memory
  *hPSF = pointer to PSF memory
  *hObject = pointer to output image memory
*/
int deconv(unsigned int nIter, size_t N1, size_t N2, size_t N3, float *hImage, float *hPSF, float *hObject){
  int ret = 0;
  hipfftResult r;
  hipError_t err;
  hipfftHandle planR2C, planC2R;

  float *im = 0;
  float *obj = 0;
  hipComplex *otf = 0;
  void *buf = 0;
  void *tmp = 0;

  size_t nSpatial = N1*N2*N3;
  size_t nFreq = N1*N2*(N3/2 + 1);
  size_t mSpatial;
  size_t mFreq;
  dim3 freqThreadsPerBlock, spatialThreadsPerBlock, freqBlocks, spatialBlocks;
  size_t tmpWork;
  err = numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = numBlocksThreads(nFreq, &freqBlocks, &freqThreadsPerBlock);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }

  mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(float);
  mFreq = freqBlocks.x * freqBlocks.y * freqBlocks.z * freqThreadsPerBlock.x * sizeof(hipComplex);

  hipDeviceReset();
  hipProfilerStart();
  // Memory Allocation
  err = hipMalloc(&im, mSpatial)
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = hipMalloc(&obj, mSpatial)
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = hipMalloc(&otf, mFreq)
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = hipMalloc(&buf, mFreq)
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = hipMemset(im, 0, mSpatial);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = hipMemset(obj, 0, mSpatial);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }

  // Memory Copy for GPU Mem
  err = hipMemcpy(im, hImage, nSpatial*sizeof(float), hipMemcpyHostToDevice);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = hipMemcpy(otf, hPSF, nSpatial*sizeof(float), hipMemcpyHostToDevice);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }
  err = hipMemcpy(obj, hObject, nSpatial*sizeof(float), hipMemcpyHostToDevice);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }

  r = createPlans(N1, N2, N3, &planR2C, &planC2R, &tmp, &tmpWork);
  if(r){
    fprintf(stderr, "CuFFT error: %d\n", r);
    return r;
  }
  r = hipfftExecR2C(planR2C, (float*)otf, otf);
  if(r){
    fprintf(stderr, "CuFFT error: %d\n", r);
    return r;
  }

  for(int i = 0; i < nIter; i++){
    r = hipfftExecR2C(planR2C, obj, (hipfftComplex*)buf);
    if(r){
      fprintf(stderr, "CuFFT error: %d\n", r);
      return r;
    }
    ComplexMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
    r = hipfftExecC2R(planC2R, (hipfftComplex*)buf, (float*)buf);
    if(r){
      fprintf(stderr, "CuFFT error: %d\n", r);
      return r;
    }
    FloatDiv<<<spatialBlocks, spatialThreadsPerBlock>>>(im, (float*)buf, (float*)buf);
    r = hipfftExecR2C(planR2C, (float*)buf, (hipfftComplex*)buf);
    if(r){
      fprintf(stderr, "CuFFT error: %d\n", r);
      return r;
    }
    ComplexMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
    r = hipfftExecC2R(planC2R, (hipfftComplex*)buf, (float*)buf);
    if(r){
      fprintf(stderr, "CuFFT error: %d\n", r);
      return r;
    }
    FloatMul<<<spatialBlocks, spatialThreadsPerBlock>>>((float*)buf, obj, obj);
  }
  // Copy output to host
  err = hipMemcpy(hObject, obj, nSpatial*sizeof(float), hipMemcpyDeviceToHost);
  if(err){
    fprintf(stderr, "CUDA error: %d\n", err);
    return err;
  }

  ret = 0;
  // Clean Up Params and Return
  if(im)
    hipFree(im);
  if(obj)
    hipFree(obj);
  if(otf)
    hipFree(otf);
  if(buf)
    hipFree(buf);
  if(tmp)
    hipFree(tmp);
  hipProfilerStop();
  hipDeviceReset();
  return ret;
}
